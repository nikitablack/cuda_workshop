#include "hip/hip_runtime.h"
#define EIGEN_NO_CUDA
#include "Eigen/Dense"
#include "utils/cuda_utils.hpp"
#include "utils/get_random.hpp"
#include "utils/math_utils.hpp"
#include "utils/Timer.hpp"

#include <cassert>

constexpr uint32_t N{2048};
constexpr uint32_t DATA_SIZE{sizeof(float) * N * N};
constexpr uint32_t BLOCK_SIZE{16};
constexpr uint32_t GRID_SIZE{N / BLOCK_SIZE};

__managed__ float ma[N * N];
__managed__ float mb[N * N];
__managed__ float mc[N * N];

__global__ void kernel()
{
  uint32_t const row{blockIdx.y * blockDim.y + threadIdx.y};
  uint32_t const col{blockIdx.x * blockDim.x + threadIdx.x};

  uint32_t offset{N * row};

  if ((row < N) && (col < N))
  {
    float result{0.0f};

    for (uint32_t s{0}; s < N; ++s)
    {
      result += ma[offset + s] * mb[col + s * N];
    }

    mc[offset + col] = result;
  }
}

int main(int argc, char *[])
{
  Eigen::MatrixXf a{N, N};
  Eigen::MatrixXf b{N, N};

  int deviceId{};
  gpuErrCheck(hipGetDevice(&deviceId));

  gpuErrCheck(hipMemPrefetchAsync(ma, DATA_SIZE, hipCpuDeviceId, nullptr));
  gpuErrCheck(hipMemPrefetchAsync(mb, DATA_SIZE, hipCpuDeviceId, nullptr));

  for (uint32_t row{0}; row < N; ++row)
  {
    for (uint32_t col{0}; col < N; ++col)
    {
      a(row, col) = ma[row * N + col] = get_random();
      b(row, col) = mb[row * N + col] = get_random();
    }
  }

  Eigen::MatrixXf c{a * b};

  gpuErrCheck(hipMemPrefetchAsync(ma, DATA_SIZE, deviceId, nullptr));
  gpuErrCheck(hipMemPrefetchAsync(mb, DATA_SIZE, deviceId, nullptr));

  for (uint32_t i{0}; i < 10; ++i)
  {
    Timer timer{};
    kernel<<<dim3{GRID_SIZE, GRID_SIZE, 1}, dim3{BLOCK_SIZE, BLOCK_SIZE, 1}>>>();
    gpuErrCheck(hipPeekAtLastError());

    gpuErrCheck(hipDeviceSynchronize());
  }

  gpuErrCheck(hipMemPrefetchAsync(mc, DATA_SIZE, hipCpuDeviceId, nullptr));

  for (uint32_t row{0}; row < N; ++row)
  {
    for (uint32_t col{0}; col < N; ++col)
    {
      assert(fuzzy_compare(mc[N * row + col], c(row, col)));
    }
  }

  return 0;
}
