#define EIGEN_NO_CUDA
#include "Eigen/Dense"
#include "utils/cuda_utils.hpp"
#include "utils/get_random.hpp"
#include "utils/math_utils.hpp"
#include "utils/Timer.hpp"

#include <cassert>

constexpr uint32_t N{2048};
constexpr uint32_t DATA_SIZE{sizeof(float) * N * N};

__managed__ float ma[N * N];
__managed__ float mb[N * N];
__managed__ float mc[N * N];

int main(int argc, char *[])
{
  Eigen::MatrixXf a{N, N};
  Eigen::MatrixXf b{N, N};

  int deviceId{};
  gpuErrCheck(hipGetDevice(&deviceId));

  gpuErrCheck(hipMemPrefetchAsync(ma, DATA_SIZE, hipCpuDeviceId, nullptr));
  gpuErrCheck(hipMemPrefetchAsync(mb, DATA_SIZE, hipCpuDeviceId, nullptr));

  for (uint32_t row{0}; row < N; ++row)
  {
    for (uint32_t col{0}; col < N; ++col)
    {
      a(row, col) = ma[row * N + col] = get_random();
      b(row, col) = mb[row * N + col] = get_random();
    }
  }

  Eigen::MatrixXf c{a * b};

  gpuErrCheck(hipMemPrefetchAsync(ma, DATA_SIZE, deviceId, nullptr));
  gpuErrCheck(hipMemPrefetchAsync(mb, DATA_SIZE, deviceId, nullptr));

  hipblasHandle_t handle;
  cublasErrCheck(hipblasCreate(&handle));

  float alpha{1.0f};
  float beta{0.0f};

  for (uint32_t i{0}; i < 10; ++i)
  {
    Timer timer{};

    cublasErrCheck(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, ma, N, mb, N, &beta, mc, N));
    gpuErrCheck(hipDeviceSynchronize());
  }

  cublasErrCheck(hipblasDestroy(handle));

  gpuErrCheck(hipMemPrefetchAsync(mc, DATA_SIZE, hipCpuDeviceId, nullptr));

  for (uint32_t row{0}; row < N; ++row)
  {
    for (uint32_t col{0}; col < N; ++col)
    {
      // if (mc[N * row + col] != c(row, col))
      // {
      //   std::cout << row << " " << col << " " << mc[N * row + col] << " " << c(row, col) << '\n';
      // }
      assert(fuzzy_compare(mc[N * row + col], c(col, row)));
    }
  }

  return 0;
}
